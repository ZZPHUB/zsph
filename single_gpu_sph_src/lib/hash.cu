#include "hip/hip_runtime.h"
#include "lib.cuh"

__global__ void cuda_ptc_hash(gpu_ptc_t *ptc_data, gpu_tmp_t *tmp_data, gpu_param_t *par)
{
    const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < par->ptc_num)
    {
        float3 pos;
        pos.x = ptc_data->pos_rho[index * 4 + 0];
        pos.y = ptc_data->pos_rho[index * 4 + 1];
        pos.z = ptc_data->pos_rho[index * 4 + 2];

        int3 hash;
        hash.x = floorf((pos.x - par->grid_xmin)/par->grid_size);
        hash.y = floorf((pos.y - par->grid_ymin)/par->grid_size); 
        hash.z = floorf((pos.z - par->grid_zmin)/par->grid_size);
        tmp_data->hash[index] = hash.z * par->grid_xdim * par->grid_ydim + hash.y * par->grid_xdim + hash.x;
        if(tmp_data->hash[index] < par->grid_hash_min || tmp_data->hash[index] > par->grid_hash_max) 
        {
            ptc_data->type[index] = 100;
            ptc_data->pos_rho[index*4+0] = par->grid_xmin;
            ptc_data->pos_rho[index*4+1] = par->grid_ymin;
            ptc_data->pos_rho[index*4+2] = par->grid_zmin;
            ptc_data->pos_rho[index*4+3] = par->rho0;
            ptc_data->vel_p[index*4+0] = 0.0f;
            ptc_data->vel_p[index*4+1] = 0.0f;
            ptc_data->vel_p[index*4+2] = 0.0f;
            ptc_data->vel_p[index*4+3] = 0.0f;
            tmp_data->hash[index] = 0;
        }
        tmp_data->index[index] = index;
    }
}