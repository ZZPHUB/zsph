#include "hip/hip_runtime.h"
#include "lib.cuh"

__global__ void cuda_ptc_hash(gpu_ptc_t *ptc_data, gpu_tmp_t *tmp_data, gpu_param_t *par)
{
    const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index < par->ptc_num)
    {
        float3 pos = ptc_data->pos[index];

        int3 hash;
        hash.x = floorf((pos.x - par->grid_xmin)/par->grid_size);
        hash.y = floorf((pos.y - par->grid_ymin)/par->grid_size); 
        hash.z = floorf((pos.z - par->grid_zmin)/par->grid_size);
        tmp_data->hash[index] = hash.z * par->grid_xdim * par->grid_ydim + hash.y * par->grid_xdim + hash.x;
        if(tmp_data->hash[index] < par->grid_hash_min || tmp_data->hash[index] > par->grid_hash_max) 
        {
            ptc_data->type[index] = 100;
            ptc_data->pos[index] = {par->grid_xmin,par->grid_ymin,par->grid_zmin};
            ptc_data->vel[index] = {0.0f,0.0f,0.0f};
            ptc_data->rhop[index] = {par->rho0,0.0f};
            tmp_data->hash[index] = 0;
        }
        tmp_data->index[index] = index;
    }
}