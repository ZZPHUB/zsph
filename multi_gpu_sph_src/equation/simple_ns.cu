#include "hip/hip_runtime.h"
#include "equation.cuh"

__global__ void cuda_boundary_ns(gpu_ptc_t *tptc_data)
{
    const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    const gpu_ptc_t ptc_data = *tptc_data;

    if(index < par.ptc_num)
    {
        float rhop_sum_tmp = 0.0f;
        float w_sum_tmp = 0.0f;
        float dofv_tmp = 0.0f;
        float3 pos = ptc_data.pos[index];
        float3 vel = ptc_data.vel[index];
        int type_0 = ptc_data.type[index];

        int type_1;
        float3 dx;
        float3 dv;
        float2 rhop_1;
        float rr2,q,w,fr;
        float dvdx;
        //int count = 0;
        for(int z=-par.grid_factor;z<=par.grid_factor;z++)
        {
            for(int y=-par.grid_factor;y<par.grid_factor;y++)
            {
                int cell_start = tmp_data.hash[index] + z * par.grid_xdim * par.grid_ydim + y * par.grid_xdim;
                int cell_end = cell_start + par.grid_factor;
                cell_start = cell_start - par.grid_factor;
                if(cell_start >= par.grid_hash_min && cell_end <= par.grid_hash_max)
                {
                    cell_start = tmp_data.grid_start[cell_start];
                    cell_end = tmp_data.grid_end[cell_end];
                    for(int i=cell_start;i<cell_end;i++)
                    {
                        if(i != index)
                        {
                            
                            dx = ptc_data.pos[i];
                            dv = ptc_data.vel[i];
                            rhop_1 = ptc_data.rhop[i];

                            dx.x = pos.x - dx.x;
                            dx.y = pos.y - dx.y;
                            dx.z = pos.z - dx.z;
                            dv.x = vel.x - dv.x;
                            dv.y = vel.y - dv.y;
                            dv.z = vel.z - dv.z;
                            rr2 = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z ;
                            dvdx = dv.x * dx.x + dv.y * dx.y + dv.z * dx.z;

                            type_1 = ptc_data.type[i];
                            rhop_1 = ptc_data.rhop[i];

                            q = sqrtf(rr2)/par.h;
                            if(q <= 2.0f)
                            {
                                //count ++;
                                fr = (1.0f - q/2.0f)*(1.0f - q/2.0f)*(1.0f - q/2.0f);
                                w = fr*(1.0f - q/2.0f);
                                w *= (2.0f*q + 1.0f)*par.adh;
                                fr *= -5.0f*par.adh/(par.h2);
                                if(type_0 != 1 && type_1 == 1)
                                {
                                    rhop_sum_tmp += (rhop_1.y - rhop_1.x*(0.0f*dx.x + 0.0f*dx.y + (0.0f - par.g)*dx.z))*w;
                                    w_sum_tmp += w;
                                }
                                else if(type_0 == 1 && type_1 == 1)
                                {
                                    dofv_tmp -= fr*dvdx*par.m/rhop_1.x;
                                }
                            }
                        }
                    }
                }
            }
        }
        
        tmp_data.dofv[index] = dofv_tmp;
        if(type_0 != 1)
        {
            if(fabs(w_sum_tmp) > 1e-8f)
            {
                rhop_sum_tmp = rhop_sum_tmp/w_sum_tmp;
            }
            else 
            {
                rhop_sum_tmp = 0.0f;
            }
            if(rhop_sum_tmp < 0.0f) rhop_sum_tmp = 0.0f;
            rhop_1.y = rhop_sum_tmp;
            rhop_1.x = rhop_sum_tmp/par.cs2 + par.rho0;
            ptc_data.rhop[index] = rhop_1;
        }
    }
}

__global__ void cuda_govering_ns(gpu_ptc_t *tptc_data)
{
    const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    const gpu_ptc_t ptc_data = *tptc_data;
    if(index < par.ptc_num)
    {
        float4 acc_drhodt = make_float4(0.0f,0.0f,0.0f,0.0f);
        float3 pos = ptc_data.pos[index];
        float3 vel = ptc_data.vel[index];
        float2 rhop_0 = ptc_data.rhop[index];
        int type_0 = ptc_data.type[index]; 
        float dofv_0 = tmp_data.dofv[index];
       
        int type_1;
        float2 rhop_1;
        float dofv_1;
        float3 dx;
        float3 dv;
        float rr2,q,fr;
        float dvdx;

        //tmp variable
        float v_tmp = 0.0f;
        
        for(int z=-par.grid_factor;z<=par.grid_factor;z++)
        {
            for(int y=-par.grid_factor;y<=par.grid_factor;y++)
            {
                int cell_start = tmp_data.hash[index] + z*par.grid_xdim*par.grid_ydim + y*par.grid_xdim;
                int cell_end = cell_start + par.grid_factor;
                cell_start = cell_start - par.grid_factor;
                if(cell_start >= par.grid_hash_min && cell_end <= par.grid_hash_max)
                {
                    cell_start = tmp_data.grid_start[cell_start];
                    cell_end = tmp_data.grid_end[cell_end];
                    for(int i=cell_start;i<cell_end;i++)
                    {
                        if(i != index )
                        {
                            dx = ptc_data.pos[i];
                            dv = ptc_data.vel[i];
                            rhop_1 = ptc_data.rhop[i];

                            type_1 = ptc_data.type[i];
                            dofv_1 = tmp_data.dofv[i];

                            //dx
                            dx.x = pos.x - dx.x;
                            dx.y = pos.y - dx.y;
                            dx.z = pos.z - dx.z;
                            
                            //dvx
                            dv.x = vel.x - dv.x;
                            dv.y = vel.y - dv.y;
                            dv.z = vel.z - dv.z;

                            rr2= dx.x*dx.x + dx.y*dx.y + dx.z*dx.z; //rr
                            dvdx = dv.x * dx.x + dv.y * dx.y + dv.z * dx.z; //dvdx
                            q = sqrtf(rr2)/par.h;

                            if(q <= 2.0f)
                            {
                                fr = -5.0f * par.adh * (1.0f - q/2.0f)*(1.0f - q/2.0f)*(1.0f - q/2.0f)/par.h2;
                                acc_drhodt.w += (rhop_0.x*dvdx*fr + (rhop_0.x-rhop_1.x)*rr2*fr*par.delta_h_cs/(rr2+par.eta2))*par.m/rhop_1.x;
                                v_tmp = -(rhop_0.y+rhop_1.y)*par.m*fr/(rhop_0.x*rhop_1.x);
                                if(type_0 == 1 && type_1 == 1)
                                {
                                    v_tmp += par.h_cs_rho_m*fr*(dofv_0 + dofv_1 + par.alpha*dvdx/(rr2+par.eta2))/(rhop_0.x*rhop_1.x);
                                }
                                acc_drhodt.x += v_tmp*dx.x;
                                acc_drhodt.y += v_tmp*dx.y;
                                acc_drhodt.z += v_tmp*dx.z;
                            }
                        }
                    }
                }
            }
        }
        acc_drhodt.z += par.g;
        tmp_data.acc_drhodt[index] = acc_drhodt;
    }
}
